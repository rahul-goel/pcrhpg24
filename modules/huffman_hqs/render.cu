#include "hip/hip_runtime.h"
#define DTABLE_SIZE 4096
#define COLOR_COMPRESSION 1 // 0 -> no compression, 1 -> bc1, 7 -> bc7

#include "huffman_kernel_data.h"
#include "hip/hip_vector_types.h"
// #include "compute_loop_las_cuda/kernel_data.h"

struct Mat {
	float4 rows[4];
};

struct BoundingBox{
	float4 position;   		//   16   0
	float4 size;       		//   16  16
	unsigned int color;     //    4  32
							// size: 48
};

struct BoundingData {
	unsigned int count;
	unsigned int instanceCount;
	unsigned int first;
	unsigned int baseInstance;
	// 16
	unsigned int pad0;
	unsigned int pad1;
	unsigned int pad2;
	unsigned int pad3;
	// 32
	unsigned int pad4;
	unsigned int pad5;
	unsigned int pad6;
	unsigned int pad7;
	// 48
	BoundingBox* ssBoxes;
};

struct ChangingRenderData{
	Mat uTransform;
	Mat uWorldView;
	Mat uProj;
	
	float3 uCamPos;
	int2 uImageSize;
	int uPointsPerThread;
	
	float3 uBoxMin;
	float3 uBoxMax;
	int uNumPoints;
	long long int uOffsetPointToData;
	int uPointFormat;
	long long int uBytesPerPoint;
	float3 uScale;
	
	int uEnableFrustumCulling;
  int showNumPoints;
  int colorizeChunks;
};

__device__ float4 matMul(const Mat& m, const float4& v)
{
	return make_float4(dot(m.rows[0], v), dot(m.rows[1], v), dot(m.rows[2], v), dot(m.rows[3], v));
}

struct Plane{
	float3 normal;
	float constant;
};

__device__ float access(float4 v, int i)
{
	if (i == 0)
		return v.x;
	if (i == 1)
		return v.y;
	if (i == 2)
		return v.z;
	return v.w;
}
__device__ float t(const ChangingRenderData& data, int index)
{
	int a = index % 4;
	int b = index / 4;
	return access(data.uTransform.rows[a], b);
}

__device__ float distanceToPoint(float3 point, Plane plane){
	return (plane.normal.x * point.x + plane.normal.y*point.y + plane.normal.z*point.z) + plane.constant;
}

__device__ Plane createPlane(float x, float y, float z, float w){

	float nLength = sqrt(x*x + y*y + z*z);
	Plane plane;
	plane.normal = make_float3(x, y, z) / nLength;
	plane.constant = w / nLength;
	return plane;
}
__device__ bool intersectsFrustum(const ChangingRenderData& d, float3 wgMin, float3 wgMax){

	Plane planes[6] = {
		createPlane(t( d,3) - t(d,0), t( d,7) - t(d,4), t(d,11) - t( d,8), t(d,15) - t(d,12)),
		createPlane(t( d,3) + t(d,0), t( d,7) + t(d,4), t(d,11) + t( d,8), t(d,15) + t(d,12)),
		createPlane(t( d,3) + t(d,1), t( d,7) + t(d,5), t(d,11) + t( d,9), t(d,15) + t(d,13)),
		createPlane(t( d,3) - t(d,1), t( d,7) - t(d,5), t(d,11) - t( d,9), t(d,15) - t(d,13)),
		createPlane(t( d,3) - t(d,2), t( d,7) - t(d,6), t(d,11) - t(d,10), t(d,15) - t(d,14)),
		createPlane(t( d,3) + t(d,2), t( d,7) + t(d,6), t(d,11) + t(d,10), t(d,15) + t(d,14)),
	};
	for(int i = 0; i < 6; i++){

		Plane plane = planes[i];

		float3 vector;
		vector.x = plane.normal.x > 0.0 ? wgMax.x : wgMin.x;
		vector.y = plane.normal.y > 0.0 ? wgMax.y : wgMin.y;
		vector.z = plane.normal.z > 0.0 ? wgMax.z : wgMin.z;

		float d = distanceToPoint(vector, plane);

		if(d < 0){
			return false;
		}
	}

	return true;
}

struct bc1_block
{
		unsigned char m_low_color[2];
		unsigned char m_high_color[2];
		unsigned char m_selectors[4];
};

__device__
unsigned int set_color(unsigned int r, unsigned int g, unsigned int b) {
  return r | (g << 8) | (b << 16);
}

__device__
unsigned int decode_bc1(int pointID, unsigned char *rgba) {
  int blockID = pointID / 16;
  int localID = pointID % 16;
  int offset = blockID * 8;

  const void* ptr = (void *) (rgba + offset);
  const bc1_block* pBlock = static_cast<const bc1_block*>(ptr);

  const unsigned int l = pBlock->m_low_color[0] | (pBlock->m_low_color[1] << 8U);
  const int cr0 = (l >> 11) & 31;
  const int cg0 = (l >> 5) & 63;
  const int cb0 = l & 31;
  const int r0 = (cr0 << 3) | (cr0 >> 2);
  const int g0 = (cg0 << 2) | (cg0 >> 4);
  const int b0 = (cb0 << 3) | (cb0 >> 2);

  const unsigned int h = pBlock->m_high_color[0] | (pBlock->m_high_color[1] << 8U);
  const int cr1 = (h >> 11) & 31;
  const int cg1 = (h >> 5) & 63;
  const int cb1 = h & 31;
  const int r1 = (cr1 << 3) | (cr1 >> 2);
  const int g1 = (cg1 << 2) | (cg1 >> 4);
  const int b1 = (cb1 << 3) | (cb1 >> 2);

  unsigned int color = -1;
  int word = (pBlock->m_selectors[localID / 4] >> (2 * (localID % 4))) & 3;
  switch (word) {
    case 0:
      color = set_color(r0, g0, b0);
      break;
    case 1:
      color = set_color(r1, g1, b1);
      break;
    case 2:
      color = set_color((r0 * 2 + r1) / 3, (g0 * 2 + g1) / 3, (b0 * 2 + b1) / 3);
      break;
    case 3:
      color = set_color((r0 + r1 * 2) / 3, (g0 + g1 * 2) / 3, (b0 + b1 * 2) / 3);
      break;
  }

  return color;
}

struct bc7_mode_6
{
  struct
  {
    unsigned long long m_mode : 7;
    unsigned long long m_r0 : 7;
    unsigned long long m_r1 : 7;
    unsigned long long m_g0 : 7;
    unsigned long long m_g1 : 7;
    unsigned long long m_b0 : 7;
    unsigned long long m_b1 : 7;
    unsigned long long m_a0 : 7;
    unsigned long long m_a1 : 7;
    unsigned long long m_p0 : 1;
  } m_lo;

  union
  {
    struct
    {
      unsigned long long m_p1 : 1;
      unsigned long long m_s00 : 3;
      unsigned long long m_s10 : 4;
      unsigned long long m_s20 : 4;
      unsigned long long m_s30 : 4;

      unsigned long long m_s01 : 4;
      unsigned long long m_s11 : 4;
      unsigned long long m_s21 : 4;
      unsigned long long m_s31 : 4;

      unsigned long long m_s02 : 4;
      unsigned long long m_s12 : 4;
      unsigned long long m_s22 : 4;
      unsigned long long m_s32 : 4;

      unsigned long long m_s03 : 4;
      unsigned long long m_s13 : 4;
      unsigned long long m_s23 : 4;
      unsigned long long m_s33 : 4;

    } m_hi;

    unsigned long long m_hi_bits;
  };
};

__device__
int linspace_idx(float start, float end, int num_points, int idx){
  float step = (end - start) / (num_points - 1);
  float val = start + idx * step;
  return round(val);
}

__device__
unsigned int decode_bc7(int pointID, unsigned char *rgba) {
  int blockID = pointID / 16;
  int localID = pointID % 16;
  int offset = blockID * 16;

  unsigned char enc[16];
  for (int i = 0; i < 16; ++i) enc[i] = rgba[offset + i];

	const bc7_mode_6 &block = *static_cast<const bc7_mode_6 *>((void *) enc);
	const unsigned int r0 = static_cast<unsigned int>((block.m_lo.m_r0 << 1) | block.m_lo.m_p0);
	const unsigned int g0 = static_cast<unsigned int>((block.m_lo.m_g0 << 1) | block.m_lo.m_p0);
	const unsigned int b0 = static_cast<unsigned int>((block.m_lo.m_b0 << 1) | block.m_lo.m_p0);
	const unsigned int a0 = static_cast<unsigned int>((block.m_lo.m_a0 << 1) | block.m_lo.m_p0);
	const unsigned int r1 = static_cast<unsigned int>((block.m_lo.m_r1 << 1) | block.m_hi.m_p1);
	const unsigned int g1 = static_cast<unsigned int>((block.m_lo.m_g1 << 1) | block.m_hi.m_p1);
	const unsigned int b1 = static_cast<unsigned int>((block.m_lo.m_b1 << 1) | block.m_hi.m_p1);
	const unsigned int a1 = static_cast<unsigned int>((block.m_lo.m_a1 << 1) | block.m_hi.m_p1);

  unsigned int color = -1;

  int idx = (block.m_hi_bits >> (localID * 4)) & 0xF;
  if (idx == 0) idx = (idx >> 1);
  const unsigned int w = linspace_idx(0, 64, 16, idx);
  const unsigned int iw = 64 - w;

  color =
  ((unsigned char) ((r0 * iw + r1 * w + 32) >> 6)) <<  0 |
  ((unsigned char) ((g0 * iw + g1 * w + 32) >> 6)) <<  8 |
  ((unsigned char) ((b0 * iw + b1 * w + 32) >> 6)) << 16 |
  ((unsigned char) ((a0 * iw + a1 * w + 32) >> 6)) << 24;

  return color;
}

__device__ void rasterize(const ChangingRenderData &data,
                          unsigned long long int *framebuffer,
                          unsigned long long int *RG,
                          unsigned long long int *BA,
                          unsigned int *Colors,
                          float3 point,
                          unsigned int index,
                          unsigned int NumPointsToRender) {
  float4 pos = matMul(data.uTransform, make_float4(point, 1.0f));
	pos.x = pos.x / pos.w;
	pos.y = pos.y / pos.w;

	float2 imgPos = {(pos.x * 0.5f + 0.5f) * data.uImageSize.x, (pos.y * 0.5f + 0.5f) * data.uImageSize.y};
	int2 pixelCoords = make_int2(imgPos.x, imgPos.y);
	int pixelID = pixelCoords.x + pixelCoords.y * data.uImageSize.x;
	unsigned int depth = *((int*)&pos.w);

	if(!(pos.w <= 0.0 || pos.x < -1 || pos.x > 1 || pos.y < -1|| pos.y > 1)){
		unsigned long long int oldPoint = framebuffer[pixelID];
    unsigned int oldDepthInt = oldPoint >> 32;
    float oldDepth = *((float*) &oldDepthInt);

    if (pos.w <= oldDepth * 1.01) {
#if COLOR_COMPRESSION==0
      unsigned int rgba = Colors[index];
#elif COLOR_COMPRESSION==1
      unsigned int rgba = decode_bc1(index, (unsigned char*) Colors);
#elif COLOR_COMPRESSION==7
      unsigned int rgba = decode_bc7(index, (unsigned char*) Colors);
#endif

      unsigned long long r = (rgba >>  0) & ((1 << 8) - 1);
      unsigned long long g = (rgba >>  8) & ((1 << 8) - 1);
      unsigned long long b = (rgba >> 16) & ((1 << 8) - 1);
      unsigned long long add;

      add = (r << 32) | g;
      atomicAdd(&RG[pixelID], add);
      add = (b << 32) | 1;
      atomicAdd(&BA[pixelID], add);
    }
	}
}

__device__
float mysmoothstep(float x) {
  const float p = 0.35f;
  const float s = 0.60f;
  const float c = 2.0f / (1.0f - s) - 1.0f;

  if (x <= p) return pow(x, c) / pow(p, c - 1.0f);
  else return 1.0f - pow(1.0f - x, c) / pow(1.0f - p, c - 1.0f);
}

extern "C" __global__
void kernel(const ChangingRenderData           cdata,
                  unsigned long long          *framebuffer,
                  unsigned long long          *RG,
                  unsigned long long          *BA,
                  GPUBatch                    *BatchData,
                  int                         *StartValues,
                  unsigned int                *EncodedData,
                  int                         *EncodedDataOffsets,
                  int                         *EncodedDataSizes,
                  int                         *SeparateData,
                  int                         *SeparateDataOffsets,
                  int                         *SeparateDataSizes,
                  int                         *DecoderTableValues,
                  int                         *DecoderTableCWLen,
                  int                         *ClusterSizes,
                  unsigned int                *Colors
                  ) {
  unsigned int batchIndex = blockIdx.x;
  unsigned int numPointsPerBatch = blockDim.x * cdata.uPointsPerThread;
  unsigned int wgFirstPoint = batchIndex * numPointsPerBatch;
  unsigned int globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int warpIdx = threadIdx.x / 32;

  // right now we dont want to deal with the edge case of last batch
	if (blockIdx.x == gridDim.x - 1) return;

  // batch meta data
  GPUBatch batch = BatchData[batchIndex];
  float3 las_offset = make_float3(batch.offset_x, batch.offset_y, batch.offset_z);
  float3 las_scale = make_float3(batch.scale_x, batch.scale_y, batch.scale_z);
  float3 las_min = make_float3(batch.las_min_x, batch.las_min_y, batch.las_min_z);


  // frustum cull if enabled
  float3 batchMin = make_float3(batch.min_x, batch.min_y, batch.min_z) - las_min;
  float3 batchMax = make_float3(batch.max_x, batch.max_y, batch.max_z) - las_min;
  if (cdata.uEnableFrustumCulling && !intersectsFrustum(cdata, batchMin, batchMax)) {
    return;
  }

  // figuring out the LOD level
  __shared__ int Shared_NumPointsToRender;
  float3 batchCenter = 0.5f * (batchMin + batchMax);
  if (threadIdx.x == 0) {
		float wgRadius = length(batchMin - batchMax);
		float4 viewCenter = matMul(cdata.uWorldView, make_float4(batchCenter, 1.0f));
		float4 viewEdge = viewCenter + make_float4(wgRadius, 0.0f, 0.0f, 0.0f);
		float4 projCenter = matMul(cdata.uProj, viewCenter);
		float4 projEdge = matMul(cdata.uProj, viewEdge);

		float2 projCenter2D = make_float2(projCenter.x, projCenter.y);
		float2 projEdge2D = make_float2(projEdge.x, projEdge.y);
		projCenter2D /= projCenter.w;
		projEdge2D /= projEdge.w;
		
		float2 screenCenter = 0.5f * (projCenter2D + 1.0f);
		screenCenter = make_float2(cdata.uImageSize.x * screenCenter.x, cdata.uImageSize.y * screenCenter.y);
		float2 screenEdge = 0.5f * (projEdge2D + 1.0f);
		screenEdge = make_float2(cdata.uImageSize.x * screenEdge.x, cdata.uImageSize.y * screenEdge.y);
		float2 diff = screenEdge - screenCenter;
		float pixelSize = sqrt(diff.x*diff.x + diff.y*diff.y);
    float percentage = 0;

    // pixelSize = min(1.0f, pixelSize * 0.01f);
    // percentage = mysmoothstep(pixelSize);
    // percentage = max(percentage, 0.1);
    // if (pixelSize > 0.8) percentage = 1.0;

    pixelSize /= 100.0;
    percentage = (1.8f * pixelSize - 0.3);
    percentage = clamp(percentage, 0.1f, 1.0f);
    percentage = 1.0;
    Shared_NumPointsToRender = min((int) (percentage * cdata.uPointsPerThread), cdata.uPointsPerThread);
    // printf("pixelSize %f percentage %f\n", pixelSize, percentage);
  }
  __syncthreads();
  int NumPointsToRender = Shared_NumPointsToRender;
  // if (threadIdx.x == 0) printf("numpointstorender %d\n", NumPointsToRender);
  // return;


  int3 prev_values = make_int3(StartValues[globalThreadIdx * 3 + 0],
                               StartValues[globalThreadIdx * 3 + 1],
                               StartValues[globalThreadIdx * 3 + 2]);

  // tracker variables for huffman
  int max_cw_size = (int) batch.max_cw_len;
  long long EncodedPtr = batch.encoding_batch_offset;
  if (warpIdx >= 1) {
    EncodedPtr += ClusterSizes[blockIdx.x * (blockDim.x / 32) + warpIdx - 1];
  }
  long long SeparatePtr = batch.separate_batch_offset;
  int sep_ptr = SeparateDataOffsets[globalThreadIdx] + SeparatePtr;

  int DCO = batch.decoder_table_offset;
  int cur_bits = 32;
  unsigned int mask = ((1 << max_cw_size) - 1) << (32 - max_cw_size);

  __shared__ int Shared_DecoderTableValues[DTABLE_SIZE];
  __shared__ char Shared_DecoderTableCWLen[DTABLE_SIZE];
  for (int i = 0; i < (1 << max_cw_size) / blockDim.x; ++i) {
    int idx = i * blockDim.x + threadIdx.x;
    Shared_DecoderTableValues[idx] = DecoderTableValues[DCO + idx];
    Shared_DecoderTableCWLen[idx] = DecoderTableCWLen[DCO + idx];
  }
  __syncthreads();


  // main loop
  int tid = threadIdx.x % 32;
  unsigned int CurHuffman = EncodedData[EncodedPtr + tid];
  unsigned int NextHuffman = EncodedData[EncodedPtr + 32 + tid];
  int already_read = 64;

  // if (threadIdx.x >= 32) return;
  // if (threadIdx.x < 32 or threadIdx.x >= 64) return;
  // if (blockIdx.x >= 1) return;
  // if (blockIdx.x <= 0 or blockIdx.x >= 2) return;
  for (int i = 0; i < NumPointsToRender; ++i) {
  // for (int i = 0; i < 20; ++i) {
    int decoded[3];
    for (int j = 0; j < 3; ++j) {
      unsigned int L = cur_bits == 32 ? CurHuffman : (CurHuffman << (32 - cur_bits));
      unsigned int R = cur_bits == 32 ? 0 : (NextHuffman >> cur_bits);
      unsigned int key = ((L|R) & mask) >> (32 - max_cw_size);

      int symbol = Shared_DecoderTableValues[key];
      int cw_size = Shared_DecoderTableCWLen[key];

      decoded[j] = (cw_size > 0 ? symbol : SeparateData[sep_ptr++]);
      cur_bits -= abs(cw_size);

      // (cur_bits <= 0) signifies whether the thread is out of bits or not
      bool need_to_read = cur_bits <= 0;
      unsigned int warp_mask = __ballot_sync(0xffffffff, need_to_read);
      // if (threadIdx.x == 0) {
      //   printf("iteration %d cur_bits %d\n", i * 3 + j, cur_bits);
      // }
      if (need_to_read) {
        int offset = __popc(warp_mask << (32 - tid));
        // int offset = __popc(warp_mask >> (32 - tid));
        CurHuffman = NextHuffman;
        NextHuffman = EncodedData[EncodedPtr + already_read + offset];
        // printf("ThreadIdx %d Idx %d warp_mask %u\n", tid, already_read + offset, warp_mask);
        cur_bits += 32;
      }
      already_read += __popc(warp_mask);
    }

    unsigned int pointIndex = wgFirstPoint + threadIdx.x * cdata.uPointsPerThread + i;
    int3 cur_values = make_int3(decoded[0] + prev_values.x,
                                decoded[1] + prev_values.y,
                                decoded[2] + prev_values.z);

    float3 cur_xyz = make_float3(cur_values.x, cur_values.y, cur_values.z) * las_scale + las_offset - las_min;
    prev_values = cur_values;

    rasterize(cdata, framebuffer, RG, BA, Colors, cur_xyz, pointIndex, NumPointsToRender);
  }
}
