#include "hip/hip_runtime.h"
#define SHARED_DTABLE
#define TRANSPOSE_ENCODED
#define DTABLE_SIZE 4096

#include "huffman_kernel_data.h"
#include "hip/hip_vector_types.h"
// #include "compute_loop_las_cuda/kernel_data.h"

struct Mat {
	float4 rows[4];
};

struct BoundingBox{
	float4 position;   		//   16   0
	float4 size;       		//   16  16
	unsigned int color;     //    4  32
							// size: 48
};

struct BoundingData {
	unsigned int count;
	unsigned int instanceCount;
	unsigned int first;
	unsigned int baseInstance;
	// 16
	unsigned int pad0;
	unsigned int pad1;
	unsigned int pad2;
	unsigned int pad3;
	// 32
	unsigned int pad4;
	unsigned int pad5;
	unsigned int pad6;
	unsigned int pad7;
	// 48
	BoundingBox* ssBoxes;
};

struct ChangingRenderData{
	Mat uTransform;
	Mat uWorldView;
	Mat uProj;
	
	float3 uCamPos;
	int2 uImageSize;
	int uPointsPerThread;
	
	float3 uBoxMin;
	float3 uBoxMax;
	int uNumPoints;
	long long int uOffsetPointToData;
	int uPointFormat;
	long long int uBytesPerPoint;
	float3 uScale;
	
	int uEnableFrustumCulling;
};

__device__ float4 matMul(const Mat& m, const float4& v)
{
	return make_float4(dot(m.rows[0], v), dot(m.rows[1], v), dot(m.rows[2], v), dot(m.rows[3], v));
}

struct Plane{
	float3 normal;
	float constant;
};

__device__ float access(float4 v, int i)
{
	if (i == 0)
		return v.x;
	if (i == 1)
		return v.y;
	if (i == 2)
		return v.z;
	return v.w;
}
__device__ float t(const ChangingRenderData& data, int index)
{
	int a = index % 4;
	int b = index / 4;
	return access(data.uTransform.rows[a], b);
}

__device__ float distanceToPoint(float3 point, Plane plane){
	return (plane.normal.x * point.x + plane.normal.y*point.y + plane.normal.z*point.z) + plane.constant;
}

__device__ Plane createPlane(float x, float y, float z, float w){

	float nLength = sqrt(x*x + y*y + z*z);
	Plane plane;
	plane.normal = make_float3(x, y, z) / nLength;
	plane.constant = w / nLength;
	return plane;
}
__device__ bool intersectsFrustum(const ChangingRenderData& d, float3 wgMin, float3 wgMax){

	Plane planes[6] = {
		createPlane(t( d,3) - t(d,0), t( d,7) - t(d,4), t(d,11) - t( d,8), t(d,15) - t(d,12)),
		createPlane(t( d,3) + t(d,0), t( d,7) + t(d,4), t(d,11) + t( d,8), t(d,15) + t(d,12)),
		createPlane(t( d,3) + t(d,1), t( d,7) + t(d,5), t(d,11) + t( d,9), t(d,15) + t(d,13)),
		createPlane(t( d,3) - t(d,1), t( d,7) - t(d,5), t(d,11) - t( d,9), t(d,15) - t(d,13)),
		createPlane(t( d,3) - t(d,2), t( d,7) - t(d,6), t(d,11) - t(d,10), t(d,15) - t(d,14)),
		createPlane(t( d,3) + t(d,2), t( d,7) + t(d,6), t(d,11) + t(d,10), t(d,15) + t(d,14)),
	};
	for(int i = 0; i < 6; i++){

		Plane plane = planes[i];

		float3 vector;
		vector.x = plane.normal.x > 0.0 ? wgMax.x : wgMin.x;
		vector.y = plane.normal.y > 0.0 ? wgMax.y : wgMin.y;
		vector.z = plane.normal.z > 0.0 ? wgMax.z : wgMin.z;

		float d = distanceToPoint(vector, plane);

		if(d < 0){
			return false;
		}
	}

	return true;
}

__device__ void rasterize(const ChangingRenderData& data, unsigned long long int* framebuffer, float3 point, unsigned int index)
{
	float4 pos = matMul(data.uTransform, make_float4(point, 1.0f));

	pos.x = pos.x / pos.w;
	pos.y = pos.y / pos.w;

	float2 imgPos = {(pos.x * 0.5f + 0.5f) * data.uImageSize.x, (pos.y * 0.5f + 0.5f) * data.uImageSize.y};
	int2 pixelCoords = make_int2(imgPos.x, imgPos.y);
	int pixelID = pixelCoords.x + pixelCoords.y * data.uImageSize.x;

	unsigned int depth = *((int*)&pos.w);
	unsigned long long int newPoint = (((unsigned long long int)depth) << 32) | index;

	if(!(pos.w <= 0.0 || pos.x < -1 || pos.x > 1 || pos.y < -1|| pos.y > 1)){
		unsigned long long int oldPoint = framebuffer[pixelID];
		if(newPoint < oldPoint){
			atomicMin(&framebuffer[pixelID], newPoint);
		}
	}
}

extern "C" __global__
void kernel(const ChangingRenderData            cdata,
                    unsigned long long          *framebuffer,
                    GPUBatch                    *BatchData,
                    int                         *StartValues,
                    unsigned int                *EncodedData,
                    int                         *EncodedDataOffsets,
                    int                         *EncodedDataSizes,
                    int                         *SeparateData,
                    int                         *SeparateDataOffsets,
                    int                         *SeparateDataSizes,
                    int                         *DecoderTableValues,
                    int                         *DecoderTableCWLen,
                    unsigned int                *Colors
                    ) {
  
  unsigned int batchIndex = blockIdx.x;
  unsigned int numPointsPerBatch = blockDim.x * cdata.uPointsPerThread;
  unsigned int wgFirstPoint = batchIndex * numPointsPerBatch;
  unsigned int globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;

  // right now we dont want to deal with the edge case of last batch
	if (blockIdx.x == gridDim.x - 1) return;

  // batch meta data
  GPUBatch batch = BatchData[batchIndex];
  float3 las_offset = make_float3(batch.offset_x, batch.offset_y, batch.offset_z);
  float3 las_scale = make_float3(batch.scale_x, batch.scale_y, batch.scale_z);
  float3 las_min = make_float3(batch.las_min_x, batch.las_min_y, batch.las_min_z);
  // printf("%d %lld\n", (int) batchIndex, batch.decoder_table_offset);


  // frustum cull if enabled
  float3 batchMin = make_float3(batch.min_x, batch.min_y, batch.min_z) - las_min;
  float3 batchMax = make_float3(batch.min_x, batch.min_y, batch.min_z) - las_min;
  if (cdata.uEnableFrustumCulling && !intersectsFrustum(cdata, batchMin, batchMax)) {
    return;
  }

  int3 prev_values = make_int3(StartValues[globalThreadIdx * 3 + 0],
                               StartValues[globalThreadIdx * 3 + 1],
                               StartValues[globalThreadIdx * 3 + 2]);


  // tracker variables for huffman
  int max_cw_size = (int) batch.max_cw_len;
  long long EncodedPtr = batch.encoding_batch_offset;
  long long SeparatePtr = batch.separate_batch_offset;

#ifdef TRANSPOSE_ENCODED
  int cur_ptr = EncodedPtr + threadIdx.x;
  int sep_ptr = SeparateDataOffsets[globalThreadIdx] + SeparatePtr;
#else
  int cur_ptr = EncodedDataOffsets[globalThreadIdx] + EncodedPtr;
  int sep_ptr = SeparateDataOffsets[globalThreadIdx] + SeparatePtr;
#endif

  int DCO = batch.decoder_table_offset;
  int cur_bits = 32;
  unsigned int mask = ((1 << max_cw_size) - 1) << (32 - max_cw_size);

#ifdef SHARED_DTABLE
  __shared__ int Shared_DecoderTableValues[DTABLE_SIZE];
  __shared__ char Shared_DecoderTableCWLen[DTABLE_SIZE];
  for (int i = 0; i < (1 << max_cw_size) / blockDim.x; ++i) {
    int idx = i * blockDim.x + threadIdx.x;
    Shared_DecoderTableValues[idx] = DecoderTableValues[DCO + idx];
    Shared_DecoderTableCWLen[idx] = DecoderTableCWLen[DCO + idx];
  }
  __syncthreads();
#endif


  // TODO - if using, correct it using EncodedPtr, SeparatePtr
  // int EDS = EncodedDataSizes[globalThreadIdx];
  // int SDS = SeparateDataSizes[globalThreadIdx];

#ifdef TRANSPOSE_ENCODED
  unsigned int CurHuffman = EncodedData[cur_ptr];
  unsigned int NextHuffman = EncodedData[cur_ptr + blockDim.x];
#else
  unsigned int CurHuffman = EncodedData[cur_ptr];
  unsigned int NextHuffman = EncodedData[cur_ptr + 1];
#endif

  for (int i = 0; i < cdata.uPointsPerThread; ++i) {
    // decode the next delta value for X, Y and Z
    int decoded[3];
    for (int j = 0; j < 3; ++j) {
      unsigned int L = cur_bits == 32 ? CurHuffman : (CurHuffman << (32 - cur_bits));
      unsigned int R = cur_bits == 32 ? 0 : (NextHuffman >> cur_bits);
      unsigned int key = ((L|R) & mask) >> (32 - max_cw_size);

#ifdef SHARED_DTABLE
      int symbol = Shared_DecoderTableValues[key];
      int cw_size = Shared_DecoderTableCWLen[key];
#else
      int DT_lookup = DCO + key;
      int symbol = DecoderTableValues[DT_lookup];
      int cw_size = DecoderTableCWLen[DT_lookup];
#endif

      decoded[j] = (cw_size > 0 ? symbol : SeparateData[sep_ptr++]);
      cur_bits -= abs(cw_size);

      // got over
      if (cur_bits <= 0) {
#ifdef TRANSPOSE_ENCODED
        cur_ptr += blockDim.x;
        CurHuffman = NextHuffman;
        NextHuffman = EncodedData[cur_ptr + blockDim.x];
#else
        cur_ptr += 1;
        CurHuffman = NextHuffman;
        NextHuffman = EncodedData[cur_ptr + 1];
#endif
        cur_bits += 32;
      }
    }

    unsigned int pointIndex = wgFirstPoint + threadIdx.x * cdata.uPointsPerThread + i;
    int3 cur_values = make_int3(decoded[0] + prev_values.x,
                                decoded[1] + prev_values.y,
                                decoded[2] + prev_values.z);

    float3 cur_xyz = make_float3(cur_values.x, cur_values.y, cur_values.z) * las_scale + las_offset - las_min;
    prev_values = cur_values;

    rasterize(cdata, framebuffer, cur_xyz, pointIndex);
  }
  return;
}
